#include "SPH_defs.h"
#include "/mnt/c/users/ondre/documents/dp/tinySPH-master/cases/heatSolid/SPH_simulationSymplectic.h"

//===================================================================//
//===========================TEST=CASES==============================//
//===================================================================//
// ========= DamBreak mDBC ======== //
//#include "../cases/damBreakLong_mDBC/SPH_simulationSymplectic.h"
//#include "../cases/damBreakLong_mDBC/SPH_simulationVerlet.h"
//#include "../cases/damBreakLong_mDBC/SPH_simulationLeapFrog.h"

// ========= DamBreak DBC ======== //
//#include "../cases/damBreakLong_DBC/SPH_simulationSymplectic.h"
//#include "../cases/damBreakLong_DBC/SPH_simulationVerlet.h"
//#include "../cases/damBreakLong_DBC/SPH_simulationLeapFrog.h"

// ========= DamBreak BT ======== //
//#include "../cases/damBreakLong_BT/SPH_simulationSymplectic.h"
//#include "../cases/damBreakLong_BT/SPH_simulationVerlet.h" /* aktualne pada */
//#include "../cases/damBreakLong_BT/SPH_simulationLeapFrog.h" /* aktualne pada */

//===================================================================//
// ====== StillWatter mDBC ======= //
//#include "../cases/stillWater_mDBC/SPH_simulationSymplectic.h"
//#include "../cases/stillWater_mDBC/SPH_simulationVerlet.h"
//#include "../cases/stillWater_mDBC/SPH_simulationLeapFrog.h"

// ====== StillWatter DBC ======= //
//#include "../cases/stillWater_DBC/SPH_simulationSymplectic.h"
//#include "../cases/stillWater_DBC/SPH_simulationVerlet.h"
//#include "../cases/stillWater_DBC/SPH_simulationLeapFrog.h"

// ====== StillWatter BT ======= //
//#include "../cases/stillWater_BT/SPH_simulationSymplectic.h"
//#include "../cases/stillWater_BT/SPH_simulationVerlet.h"

//===================================================================//
// ======= flowDrop mDBC ======== //
//#include "../cases/flowDrop_mDBC/SPH_simulationSymplectic.h"
//#include "../cases/flowDrop_mDBC/SPH_simulationLeapFrog.h"

//===================================================================//
//=======================EXPERIMENTAL=CASES===========================//
//===================================================================//
// ==== DamBreak mDBC-reinitialization === //
//#include "../cases/damBreakLong_mDBC/SPH_simulationSymplectic_reinit.h"
//#include "../cases/damBreakLong_mDBC/SPH_simulationVerlet_reinit.h"

// ========= DamBreak GWBC ======== //
//#include "../cases/damBreakLong_GWBC/SPH_simulation.h" /* zatim stale nefunguje */

//===================================================================//

// --- Still Watter BT --- //
//#include "../cases/stillWater_BT/SPH_simulationSymplectic.h"

// --- RALESPH Still Watter --- //
//#include "../cases/ALESPH_stillWater/MUSCL.h"

// --- DamBreak mDBC --- //
//#include "../cases/damBreak_mDBC/SPH_simulationSymplectic.h"

// --- DamBreak BT --- //
//#include "../cases/damBreak_BT/SPH_simulationSymplectic.h"

// --- RSPH DamBreak --- //
//#include "../cases/RSPH_dambreak/SPH_simulation.h"
//#include "../cases/RSPH_dambreak/SPH_simulationSymplectic.h"

// --- RALESPH DamBreak --- //
//#include "../cases/ALESPH_dambreak/MUSCL.h"
//#include "../cases/ALESPH_dambreak/SPH_simulationINT.h"

// --- inflowSlab mDBC --- //
//#include "../cases/inflowSlab_mDBC/SPH_simulationSymplectic.h"
//tohle zatim nefunguje: #include "../cases/inflowSlab_mDBC/SPH_simulationVerlet.h"

//#include "../cases/damBreakLong_BT/SPH_simulationSymplectic.h"
//#include "../cases/damBreakLong_BToprava/SPH_simulationSymplectic.h"
//#include "../cases/ALESPH_dambreakLong/MUSCL.h"

//ALESOON
//===================================================================//
//===========================IN=PROGRESS=============================//
//===================================================================//
//#include "../cases/ALESPH_dambreakLongNEW/SPH_simulationINT.h"
//#include "../cases/ALESPH_dambreakLongNEW/SPH_simulationINTVerlet.h"
//#include "../cases/RSPH_damBreakLong/SPH_simulationSymplectic.h"
//===================================================================//


int main(){

	printf("working... \n");

	SPH_simulation SIMULATION;

	SIMULATION.PREP_SIMULATION_DATA(); //move to simulation
	SIMULATION.INIT();
	SIMULATION.RUN();

	return EXIT_SUCCESS;
}
